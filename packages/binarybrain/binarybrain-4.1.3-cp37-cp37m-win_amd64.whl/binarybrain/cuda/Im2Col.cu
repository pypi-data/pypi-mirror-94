#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <algorithm>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



//////////////////////////////
// forward
//////////////////////////////

#define IM2COL_BORDER_CONSTANT      0
#define IM2COL_BORDER_REFLECT       1
#define IM2COL_BORDER_REFLECT_101   2
#define IM2COL_BORDER_REPLICATE     3
#define IM2COL_BORDER_WRAP          4

__device__ __forceinline__ bool device_Im2Col_Border(int mode, int &x, int &y, int w, int h)
{
    switch ( mode ) {
    case IM2COL_BORDER_REFLECT:
        if ( x < 0  ) { x = -x - 1; }
        if ( y < 0  ) { y = -y - 1; }
        if ( x >= w ) { x = (w - 1) - (x - w); }
        if ( y >= h ) { y = (h - 1) - (y - h); }
        return true;
    
    case IM2COL_BORDER_REFLECT_101:
        if ( x < 0  ) { x = -x; }
        if ( y < 0  ) { y = -y; }
        if ( x >= w ) { x = (w - 2) - (x - w); }
        if ( y >= h ) { y = (h - 2) - (y - h); }
        return true;

    case IM2COL_BORDER_REPLICATE:
        if ( x < 0  ) { x = 0; }
        if ( y < 0  ) { y = 0; }
        if ( x >= w ) { x = w - 1; }
        if ( y >= h ) { y = h - 1; }
        return true;

    case IM2COL_BORDER_WRAP:
        if ( x < 0  ) { x += w; }
        if ( y < 0  ) { y += h; }
        if ( x >= w ) { x -= w; }
        if ( y >= h ) { y -= h; }
        return true;
    }

    return false;
}



__global__ void kernal_fp32_Im2Col_Forward(
            float const     *x_buf,
            float           *y_buf,
            int             x_stride,
            int             y_stride,
            int             x_offset,
            int             y_offset,
            int             input_frame_stride,
            int             input_w_size,
            int             input_h_size,
            int             output_frame_size,
            int             output_frame_stride,
            int             output_w_size,
            int             output_size,
            int             border_mode,
            float           border_value
        )
{
    int filter_w_size = blockDim.y;
    int filter_h_size = blockDim.z;

    int output_frame = blockDim.x * blockIdx.x + threadIdx.x;

    if ( output_frame < output_frame_size ) {
        int fx           = threadIdx.y;
        int fy           = threadIdx.z;
        int c            = blockIdx.y;
    
        int input_frame = output_frame / output_size;
        int f           = output_frame % output_size;
        int iy = (f / output_w_size) * y_stride - y_offset + fy;
        int ix = (f % output_w_size) * x_stride - x_offset + fx;

        float x = border_value;
        if ( iy >= 0 && iy < input_h_size && ix >= 0 && ix < input_w_size ) {
            int input_node  = (c * input_h_size  + iy) * input_w_size  + ix;
            x = x_buf[input_node * input_frame_stride + input_frame];
        }
        else {
            if ( device_Im2Col_Border(border_mode, ix, iy, input_w_size, input_h_size) ) {
                int input_node  = (c * input_h_size  + iy) * input_w_size  + ix;
                x = x_buf[input_node * input_frame_stride + input_frame];
            }
        }

        int output_node = (c * filter_h_size + fy) * filter_w_size + fx;    
        y_buf[output_node * output_frame_stride + output_frame] = x;
    }
}


BBCU_DLL_EXPORT int bbcu_fp32_Im2Col_Forward
        (
            float const     *dev_x_buf,
            float           *dev_y_buf,
            int             x_stride,
            int             y_stride,
            int             x_offset,
            int             y_offset,
            int             input_frame_size,
            int             input_frame_stride,
            int             input_w_size,
            int             input_h_size,
            int             input_c_size,
            int             output_w_size,
            int             output_h_size,
            int             output_frame_stride,
            int             filter_w_size,
            int             filter_h_size,
            int             border_mode,
            float           border_value,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    int output_c_size = input_c_size;
    int output_size   = output_w_size * output_h_size;
    
    int output_frame_size = input_frame_size * output_size;
    
    int     frame_unit = 1024;
    while ( frame_unit * filter_w_size * filter_h_size > 1024 ) { frame_unit /= 2; }
    BBCU_ASSERT(frame_unit > 0);

    dim3    block(frame_unit, filter_w_size, filter_h_size);
    dim3    grid((output_frame_size + (frame_unit-1))/frame_unit, output_c_size);
    
    kernal_fp32_Im2Col_Forward<<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            x_stride,
            y_stride,
            x_offset,
            y_offset,
            input_frame_stride,
            input_w_size,
            input_h_size,          
            output_frame_size,
            output_frame_stride,
            output_w_size,
            output_size,
            border_mode,
            border_value
        );
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}


__global__ void kernal_bit_Im2Col_Forward(
            int const       *x_buf,
            int             *y_buf,
            int             x_stride,
            int             y_stride,
            int             x_offset,
            int             y_offset,
            int             input_frame_stride,
            int             input_w_size,
            int             input_h_size,           
            int             output_frame_size,
            int             output_frame_stride,
            int             output_w_size,
            int             output_size,
            int             border_mode
    )
{
    int output_frame_unit = blockDim.x * blockIdx.x + threadIdx.x;

    if ( output_frame_unit < output_frame_stride ) {
        int filter_w_size = blockDim.y;
        int filter_h_size = blockDim.z;

        int fx          = threadIdx.y;
        int fy          = threadIdx.z;
        int c           = blockIdx.y;

        int output_node = (c * filter_h_size + fy) * filter_w_size + fx;

        int y = 0;
        for ( int i = 0; i < 32; ++i ) {
            int output_frame = output_frame_unit * 32 + i;
            if ( output_frame < output_frame_size ) {
                int input_frame = output_frame / output_size;
                int f           = output_frame % output_size;
                int iy = (f / output_w_size) * y_stride - y_offset + fy;
                int ix = (f % output_w_size) * x_stride - x_offset + fx;

                if ( iy >= 0 && iy < input_h_size && ix >= 0 && ix < input_w_size ) {
                    int input_node  = (c * input_h_size  + iy) * input_w_size  + ix;
                    int const *x_ptr = &x_buf[input_node  * input_frame_stride];
                    int x = ((x_ptr[input_frame / 32] >> (input_frame % 32)) & 1);
                    y |= (x << i);
                }
                else {
                    if ( device_Im2Col_Border(border_mode, ix, iy, input_w_size, input_h_size) ) {
                        int input_node  = (c * input_h_size  + iy) * input_w_size  + ix;
                        int const *x_ptr = &x_buf[input_node  * input_frame_stride];
                        int x = ((x_ptr[input_frame / 32] >> (input_frame % 32)) & 1);
                        y |= (x << i);
                    }
                }
            }
        }

        int *y_ptr = &y_buf[output_node * output_frame_stride];
        y_ptr[output_frame_unit] = y;
    }
}

BBCU_DLL_EXPORT int bbcu_bit_Im2Col_Forward
        (
            int const       *dev_x_buf,
            int             *dev_y_buf,
            int             x_stride,
            int             y_stride,
            int             x_offset,
            int             y_offset,
            int             input_frame_size,
            int             input_frame_stride,
            int             input_w_size,
            int             input_h_size,
            int             input_c_size,
            int             output_w_size,
            int             output_h_size,
            int             output_frame_stride,
            int             filter_w_size,
            int             filter_h_size,
            int             border_mode,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    int output_c_size = input_c_size;
    int output_size   = output_w_size * output_h_size;
    
    int output_frame_size = input_frame_size * output_size;
    int output_frame_unit = (output_frame_size + 31) / 32;

    int     frame_unit = 16;
    dim3    grid((output_frame_unit + (frame_unit-1))/frame_unit, output_c_size);
    dim3    block(frame_unit, filter_w_size, filter_h_size);
    
    kernal_bit_Im2Col_Forward<<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            x_stride,
            y_stride,
            x_offset,
            y_offset,
            input_frame_stride,
            input_w_size,
            input_h_size,
            output_frame_size,
            output_frame_stride,
            output_w_size,
            output_size,
            border_mode
        );
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}




//////////////////////////////
// backward
//////////////////////////////

__global__ void kernal_fp32_Im2Col_Backward(
            float const     *dy_buf,
            float           *dx_buf,

            int             x_stride,
            int             y_stride,
            int             x_offset,
            int             y_offset,

            int             input_frame_size,
            int             input_frame_stride,
            int             input_w_size,
            int             input_h_size,
            int             input_c_size,
            
            int             output_frame_size,
            int             output_frame_stride,
            int             output_w_size,
            int             output_h_size,

            int             filter_w_size,
            int             filter_h_size
        )
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int c = blockDim.z * blockIdx.z + threadIdx.z;
    
    if ( x < input_w_size && y < input_h_size && c < input_c_size ) {
        float const *dy_ptr = &dy_buf[c * filter_h_size * filter_w_size * output_frame_stride];

        int iy_limit = (output_h_size - 1) * y_stride;
        int ix_limit = (output_w_size - 1) * x_stride;

        int x_align = x % x_stride;
        int y_align = y % y_stride;

        for ( int input_frame = 0; input_frame < input_frame_size; ++input_frame ) {
            float dx = 0;
            for (int fy = y_align; fy < filter_h_size; fy += y_stride ) {
                int iy = y - fy + y_offset;
                if ( iy >= 0 && iy <= iy_limit ) {
                    for (int fx = x_align; fx < filter_w_size; fx += x_stride) {
                        int ix = x - fx + x_offset;
                        if (ix >= 0 && ix <= ix_limit ) {
                            int output_frame = (input_frame * output_h_size + (iy/y_stride)) * output_w_size + (ix/x_stride);
                            int output_node  = fy * filter_w_size + fx;
                            dx += dy_ptr[output_node * output_frame_stride + output_frame];
                        }
                    }
                }
            }
            dx_buf[((c * input_h_size + y) * input_w_size + x) * input_frame_stride + input_frame] = dx;
        }
    }
}



BBCU_DLL_EXPORT int bbcu_fp32_Im2Col_Backward
        (
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            int             x_stride,
            int             y_stride,
            int             x_offset,
            int             y_offset,
            int             input_frame_size,
            int             input_frame_stride,
            int             input_w_size,
            int             input_h_size,
            int             input_c_size,
            int             output_w_size,
            int             output_h_size,
            int             output_frame_stride,            
            int             filter_w_size,
            int             filter_h_size,            
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

//  int output_c_size = input_c_size;
//    int output_w_size = input_w_size - filter_w_size + 1;
//    int output_h_size = input_h_size - filter_h_size + 1;
    int output_size   = output_w_size * output_h_size;
    
    int output_frame_size = input_frame_size * output_size;
    
    dim3    block(1024, 1, 1);
    while ( (int)block.x / 2 >= input_w_size ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= input_h_size ) { block.y /= 2; block.z *= 2; }
    while ( (int)block.z / 2 >= input_c_size ) { block.z /= 2; }
    block.z = std::min(64, (int)block.z);

    dim3    grid;
    grid.x = (input_w_size + block.x - 1) / block.x;
    grid.y = (input_h_size + block.y - 1) / block.y;
    grid.z = (input_c_size + block.z - 1) / block.z;

//  dim3    grid(input_w_size, input_h_size, 1);
//  dim3    block(1, 1, input_c_size);
    
    kernal_fp32_Im2Col_Backward<<<grid, block, 0, streamId>>>(
            dev_dy_buf,
            dev_dx_buf,
            x_stride,
            y_stride,
            x_offset,
            y_offset,
            input_frame_size,
            input_frame_stride,
            input_w_size,
            input_h_size,
            input_c_size,
            output_frame_size,
            output_frame_stride,
            output_w_size,
            output_h_size,
            filter_w_size,
            filter_h_size
        );
    BB_CUDA_CHECK_LAST_ERROR();

    return 0;
}



