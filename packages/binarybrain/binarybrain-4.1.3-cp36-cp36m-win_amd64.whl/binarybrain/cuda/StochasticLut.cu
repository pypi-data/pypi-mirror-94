#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <chrono>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"
#include "Common.cuh"
#include "StochasticLut.cuh"


// -------------------------------------------------
//  Forward
// -------------------------------------------------

// real type
template<int N=6, typename T=float, int MAX_NODE_UNIT=32>
__global__ void kernal_StochasticLut_Forward(
            T   const   *x_buf,
            T           *y_buf,
            int const   *input_index,
            T   const   *W_buf,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            int         input_binary,
            int         lut_binarize,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  T       W[(1<<N)][MAX_NODE_UNIT];
                T const *x_ptr[N];
                T       *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < (1<<N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1<<N) + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < N; ++i ) {
            x_ptr[i] = &x_buf[frame_stride * input_index[N*node + i]];
        }

        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    for (int frame = id; frame < frame_size; frame += id_step) {
        if ( node < node_size ) {
            T   x[N];
            if ( input_binary ) {
                for ( int i = 0; i < N; ++i) {
                    x[i] = 0.5 + ((x_ptr[i][frame] > 0.5) ? +unbinarize_bias : -unbinarize_bias);
                }
            }
            else {
                for ( int i = 0; i < N; ++i) {
                    x[i] = min(1.0, max(0.0, x_ptr[i][frame]));
                }
            }

            T   y = StochasticLut<N, T, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

            // clamp
            y = max(0.0, y);
            y = min(1.0, y);
        
            y_ptr[frame] = y;
        }
    }
}


template <int N>
BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Forward
        (
            const float     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             input_binary,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 512;
    unsigned int const MAX_NODE_UNIT  = 64;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_StochasticLut_Forward<N, float, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            node_size,
            frame_size,
            frame_stride,
            input_binary,
            lut_binarize,
            unbinarize_bias
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}



// bit packing
template<int N=6, typename T=float, int MAX_NODE_UNIT=32>
__global__ void kernal_bit_StochasticLut_Forward(
            int const   *x_buf,
            T           *y_buf,
            int const   *input_index,
            T   const   *W_buf,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            int         bin_frame_stride,
            int         binary_mode,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__ T    W[(1 << N)][MAX_NODE_UNIT];
    int   const     *x_ptr[N];
    T               *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < (1 << N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1 << N) + i];
            if ( binary_mode ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < N; ++i ) {
            x_ptr[i] = &x_buf[bin_frame_stride * input_index[N*node + i]];
        }

        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    for (int frame = id; frame < frame_size; frame += id_step) {
        if ( node < node_size ) {
            int bit_mask = (1 << (frame & 0x1f));
            int unit     = (frame >> 5);
            
            // read x
            T   x[N];
            for ( int i = 0; i < N; ++i) {
                x[i] = 0.5 + ((x_ptr[i][unit] & bit_mask) ? +unbinarize_bias : -unbinarize_bias);
            }

            // calculate
            T   y = StochasticLut<N, T, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

            // clamp
            y = max(0.0, y);
            y = min(1.0, y);

            y_ptr[frame] = y;
        }
    }
}


template <int N>
BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Forward
        (
            int   const     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 512;
    unsigned int const MAX_NODE_UNIT  = 64;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  ) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_bit_StochasticLut_Forward<N, float, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            node_size,
            frame_size,
            frame_stride,
            bin_frame_stride,
            lut_binarize,
            unbinarize_bias
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}



// bit packing and binarize
template<int N=6, typename T=float, int MAX_NODE_UNIT=32>
__global__ void kernal_bit_bit_StochasticLut_Forward(
            int const   *x_buf,
            int         *y_buf,
            int const   *input_index,
            T   const   *W_buf,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            int         binary_mode,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__ T    W[(1 << N)][MAX_NODE_UNIT];
    int   const     *x_ptr[N];
    int             *y_ptr;
    
    if ( node < node_size ) {
        // read W
        for ( int i = id; i < (1 << N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1 << N) + i];
            if ( binary_mode ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < N; ++i ) {
            x_ptr[i] = &x_buf[frame_stride * input_index[N*node + i]];
        }

        y_ptr = &y_buf[node * frame_stride];
    }

    __syncthreads();
    
    T   unbinarize_hi = 0.5 + unbinarize_bias;
    T   unbinarize_lo = 0.5 - unbinarize_bias;

    if ( node < node_size ) {
        int frame_unit_size = ((frame_size + 0x1f) & ~0x1f);
        for (int frame = id; frame < frame_unit_size; frame += id_step) {
            int y_mask   = 0;
            int unit     = (frame >> 5);
            int bit      = (frame & 0x1f);
            int bit_mask = (1 << bit);
            if ( frame < frame_size ) {
                // read x
                T   x[N];
                for ( int i = 0; i < N; ++i) {
                    x[i] = ((x_ptr[i][unit] & bit_mask) ? unbinarize_hi : unbinarize_lo);
                }

                // calculate
                T   y = StochasticLut<N, T, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

                // binarize
                if ( y > 0.5 ) {
                    y_mask = bit_mask;
                }
            }

            // OR
            y_mask = device_int_ShuffleOr(y_mask);

            if ( bit == 0 ) {
                y_ptr[unit] = y_mask;
            }
        }
    }
}


template <int N>
BBCU_DLL_EXPORT int bbcu_bit_bit_fp32_StochasticLut_Forward
        (
            int   const     *dev_x_buf,
            int             *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_W,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 512;
    unsigned int const MAX_NODE_UNIT  = THREAD_SIZE / 32;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size && block.x > 32 ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size                  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size                  ) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size && block.x > 32 ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_bit_bit_StochasticLut_Forward<N, float, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            node_size,
            frame_size,
            frame_stride,
            lut_binarize,
            unbinarize_bias
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}



// -------------------------------------------------
//  Backward
// -------------------------------------------------

// real type
template<int N=6, typename T=float, int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_StochasticLut_Backward
        (
            T   const   *x_buf,
            T   const   *dy_buf,
            T           *dx_buf,
            int const   *input_index,
            T   const   *W_buf,
            T           *dW_buf,
            int         node_size,
            int         frame_size,
            int         frame_stride,
            int         dx_frame_stride,
            int         input_binary,
            int         lut_binarize,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  T       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  T       dW_prev[(1 << N)][MAX_NODE_UNIT];
    __shared__  T       W[(1 << N)][MAX_NODE_UNIT];
                T       dW[(1 << N)];
                T const *x_ptr[N];
                T const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        for ( int i = 0; i < (1 << N); ++i) {
            dW[i] = 0;
        }

        for ( int i = id; i < (1 << N); i += id_step ) {
            dW_prev[i][node_id] = dW_buf[node * (1 << N) + i];
        }

        // read W
        for ( int i = id; i < (1 << N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1 << N) + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
    
        // init pointer
        for ( int i = 0; i < N; ++i ) {
            int input_node = input_index[N*node + i];
            x_ptr[i]  = &x_buf[input_node * frame_stride];
        }

        dy_ptr = &dy_buf[node * frame_stride];
    }

    __syncthreads();

    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            // read x
            T   x[N];
            if ( input_binary ) {
                for ( int i = 0; i < N; ++i) {
                    x[i] = 0.5 +((x_ptr[i][frame] > 0.5)  ? +unbinarize_bias : -unbinarize_bias);
                }
            }
            else {
                for ( int i = 0; i < N; ++i) {
                    x[i] = max(0.0, min(1.0, x_ptr[i][frame]));
                }
            }

            // read dy
            T   dy = dy_ptr[frame];

            // calculate
            StochasticLut<N, T, MAX_NODE_UNIT>::NodeBackward(node_id, x, dy, &dx_buf[node*N*dx_frame_stride + frame], W, dW, dx_frame_stride);
        }
    }

    for ( int i = 0; i < (1 << N); ++i ) {
        dW[i] = device_fp32_LocalSum(dW[i], sbuf[node_id]);
    }

    if ( node < node_size ) {
        if ( id == 0 ) {
            for ( int i = 0; i < (1 << N); ++i) {
                dW_buf[node*(1 << N) + i] = dW[i] + dW_prev[i][node_id];
            }
        }
    }
}


template <int N>
BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Backward
        (
            float const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            int   const     *dev_reverse_index,
            float const     *dev_W,
            float           *dev_dW,
            int             reverse_index_stride,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             tmp_frame_size,
            int             tmp_frame_stride,
            int             input_binary,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    int frame_offset = 0;
    do {
        int unit_frame_size = frame_size - frame_offset;
        if (unit_frame_size > tmp_frame_size) {
            unit_frame_size = tmp_frame_size;
        }

        {
            unsigned int const THREAD_SIZE    = 256;
            unsigned int const MAX_FRAME_UNIT = 256;
            unsigned int const MAX_NODE_UNIT  = 16;

    #if 0
            dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
            while ( (int)block.x / 2 >= unit_frame_size  ) { block.x /= 2; block.y *= 2; }
            while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
    #else
            dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
            while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; block.x *= 2;}
            while ( (int)block.x / 2 >= unit_frame_size  ) { block.x /= 2; }
    #endif

            block.x = std::min(block.x, MAX_FRAME_UNIT);
            block.y = std::min(block.y, MAX_NODE_UNIT);
            dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

            kernal_StochasticLut_Backward<N, float, MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
                    dev_x_buf  + frame_offset,
                    dev_dy_buf + frame_offset,
                    dev_dx_tmp,
                    dev_input_index,
                    dev_W,
                    dev_dW,
                    output_node_size,
                    unit_frame_size,
                    frame_stride,
                    tmp_frame_stride,
                    input_binary,
                    lut_binarize,
                    unbinarize_bias
                );
            BB_CUDA_CHECK_LAST_ERROR();
        }
    

        {
            unsigned int const THREAD_SIZE    = 1024;
            unsigned int const MAX_FRAME_UNIT = 1024;
            unsigned int const MAX_NODE_UNIT  = 1024;

    #if 1
            dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
            while ( (int)block.x / 2 >= unit_frame_size ) { block.x /= 2; block.y *= 2; }
            while ( (int)block.y / 2 >= input_node_size ) { block.y /= 2; }
    #else
            dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
            while ( (int)block.y / 2 >= input_node_size ) { block.y /= 2; block.x *= 2;}
            while ( (int)block.x / 2 >= unit_frame_size ) { block.x /= 2; }
    #endif

            block.x = std::min(block.x, MAX_FRAME_UNIT);
            block.y = std::min(block.y, MAX_NODE_UNIT);
            dim3    grid((unit_frame_size + (block.x - 1)) / block.x, (input_node_size + (block.y - 1)) / block.y);

            kernal_NodeIntegrateWithTable<float><<<grid, block>>>
                (
                    dev_dx_tmp,
                    dev_dx_buf + frame_offset,
                    dev_reverse_index,
                    reverse_index_stride,
                    input_node_size,
                    unit_frame_size,
                    tmp_frame_stride,
                    frame_stride
                );
            BB_CUDA_CHECK_LAST_ERROR();
        }

        frame_offset += unit_frame_size;
    } while ( frame_offset < frame_size );

    return 0;
}


// bit packing
template<int N=6, typename T=float, int MAX_FRAME_UNIT=256, int MAX_NODE_UNIT=16>
__global__ void kernal_bit_StochasticLut_Backward
        (
            int const   *x_buf,
            T   const   *dy_buf,
            T           *dx_buf,
            int const   *input_index,
            T   const   *W_buf,
            T           *dW_buf,
            int         node_size,
            int         frame_size,
            int         x_frame_stride,
            int         dy_frame_stride,
            int         dx_frame_stride,
            int         lut_binarize,
            T           unbinarize_bias
        )
{
    int node_id = threadIdx.y;
    int node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id      = threadIdx.x;
    int id_step = blockDim.x;

    __shared__  T       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  T       dW_prev[(1 << N)][MAX_NODE_UNIT];
    __shared__  T       W[(1 << N)][MAX_NODE_UNIT];
                T       dW[(1 << N)];
                int   const *x_ptr[N];
                T const *dy_ptr;
    
    // initialize dW
    if ( node < node_size ) {
        for ( int i = 0; i < (1 << N); ++i) {
            dW[i] = 0;
        }

        for ( int i = id; i < (1 << N); i += id_step ) {
            dW_prev[i][node_id] = dW_buf[node * (1 << N) + i];
        }

        // read W
        for ( int i = id; i < (1 << N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1 << N) + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > 0.5 ? 1.0 : 0.0;
            }
        }
        
        // init pointer
        for ( int i = 0; i < N; ++i ) {
            int input_node = input_index[N*node + i];
            x_ptr[i]  = &x_buf[input_node * x_frame_stride];
        }

        dy_ptr = &dy_buf[node * dy_frame_stride];
    }

    __syncthreads();

    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);

            // read x
            T   x[N];
            for ( int i = 0; i < N; ++i) {
                x[i] = 0.5 +((x_ptr[i][unit] & bit) ? +unbinarize_bias : -unbinarize_bias);
            }

            // read dy
            T   dy = dy_ptr[frame];

            // calculate
            StochasticLut<N, T, MAX_NODE_UNIT>::NodeBackward(node_id, x, dy, &dx_buf[node*N*dx_frame_stride + frame], W, dW, dx_frame_stride);
        }
    }

    // write dW
    for ( int i = 0; i < (1 << N); ++i ) {
        dW[i] = device_fp32_LocalSum(dW[i], sbuf[node_id]);
    }

    if ( node < node_size ) {
        if ( id == 0 ) {
            for ( int i = 0; i < (1 << N); ++i) {
                dW_buf[node*(1 << N) + i] = dW[i] + dW_prev[i][node_id];
            }
        }
    }
}


template <int N>
BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Backward
        (
            int   const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            int   const     *dev_reverse_index,
            float const     *dev_W,
            float           *dev_dW,
            int             reverse_index_stride,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            int             bin_frame_stride,
            int             tmp_frame_size,
            int             tmp_frame_stride,
            int             lut_binarize,
            float           unbinarize_bias,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    int frame_offset = 0;
    do {
        int unit_frame_size = frame_size - frame_offset;
        if (unit_frame_size > tmp_frame_size) {
            unit_frame_size = tmp_frame_size;
        }

        {
            unsigned int const THREAD_SIZE    = 256;
            unsigned int const MAX_FRAME_UNIT = 256;
            unsigned int const MAX_NODE_UNIT  = 16;

    #if 0
            dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
            while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
            while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
    #else
            dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
            while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
            while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
    #endif

            block.x = std::min(block.x, MAX_FRAME_UNIT);
            block.y = std::min(block.y, MAX_NODE_UNIT);
            dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

            kernal_bit_StochasticLut_Backward<N, float, MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
                    dev_x_buf  + (frame_offset / 32),
                    dev_dy_buf + frame_offset,
                    dev_dx_tmp,
                    dev_input_index,
                    dev_W,
                    dev_dW,
                    output_node_size,
                    unit_frame_size,
                    bin_frame_stride,
                    frame_stride,
                    tmp_frame_stride,
                    lut_binarize,
                    unbinarize_bias
                );
            BB_CUDA_CHECK_LAST_ERROR();
        }
    

        {
            unsigned int const THREAD_SIZE    = 1024;
            unsigned int const MAX_FRAME_UNIT = 1024;
            unsigned int const MAX_NODE_UNIT  = 1024;

    #if 1
            dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
            while ( (int)block.x / 2 >= unit_frame_size ) { block.x /= 2; block.y *= 2; }
            while ( (int)block.y / 2 >= input_node_size ) { block.y /= 2; }
    #else
            dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
            while ( (int)block.y / 2 >= input_node_size ) { block.y /= 2; block.x *= 2;}
            while ( (int)block.x / 2 >= unit_frame_size ) { block.x /= 2; }
    #endif

            block.x = std::min(block.x, MAX_FRAME_UNIT);
            block.y = std::min(block.y, MAX_NODE_UNIT);
            dim3    grid((unit_frame_size + (block.x - 1)) / block.x, (input_node_size + (block.y - 1)) / block.y);

            kernal_NodeIntegrateWithTable<float><<<grid, block>>>
                (
                    dev_dx_tmp,
                    dev_dx_buf + frame_offset,
                    dev_reverse_index,
                    reverse_index_stride,
                    input_node_size,
                    unit_frame_size,
                    tmp_frame_stride,
                    frame_stride
                );
            BB_CUDA_CHECK_LAST_ERROR();
        }

        frame_offset += unit_frame_size;
    } while ( frame_offset < frame_size );
    

    return 0;
}



// 実体化
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Forward<6>(const float *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Forward<5>(const float *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Forward<4>(const float *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Forward<3>(const float *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Forward<2>(const float *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);

template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Forward<6>(int const *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int bin_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Forward<5>(int const *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int bin_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Forward<4>(int const *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int bin_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Forward<3>(int const *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int bin_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Forward<2>(int const *dev_x_buf, float *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int bin_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);

template BBCU_DLL_EXPORT int bbcu_bit_bit_fp32_StochasticLut_Forward<6>(int const *dev_x_buf, int *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_bit_fp32_StochasticLut_Forward<5>(int const *dev_x_buf, int *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_bit_fp32_StochasticLut_Forward<4>(int const *dev_x_buf, int *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_bit_fp32_StochasticLut_Forward<3>(int const *dev_x_buf, int *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_bit_fp32_StochasticLut_Forward<2>(int const *dev_x_buf, int *dev_y_buf, int const *dev_input_index, float const *dev_W, int node_size, int frame_size, int frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);

template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Backward<6>(float const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int tmp_frame_size, int tmp_frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Backward<5>(float const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int tmp_frame_size, int tmp_frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Backward<4>(float const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int tmp_frame_size, int tmp_frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Backward<3>(float const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int tmp_frame_size, int tmp_frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_fp32_StochasticLut_Backward<2>(float const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int tmp_frame_size, int tmp_frame_stride, int input_binary, int lut_binarize, float unbinarize_bias, hipStream_t streamId);

template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Backward<6>(int const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int bin_frame_stride, int tmp_frame_size, int tmp_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Backward<5>(int const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int bin_frame_stride, int tmp_frame_size, int tmp_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Backward<4>(int const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int bin_frame_stride, int tmp_frame_size, int tmp_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Backward<3>(int const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int bin_frame_stride, int tmp_frame_size, int tmp_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);
template BBCU_DLL_EXPORT int bbcu_bit_fp32_StochasticLut_Backward<2>(int const *dev_x_buf, float const *dev_dy_buf, float *dev_dx_buf, float *dev_dx_tmp, int const *dev_input_index, int const *dev_reverse_index, float const *dev_W, float *dev_dW, int reverse_index_stride, int input_node_size, int output_node_size, int frame_size, int frame_stride, int bin_frame_stride, int tmp_frame_size, int tmp_frame_stride, int lut_binarize, float unbinarize_bias, hipStream_t streamId);


// end of file
