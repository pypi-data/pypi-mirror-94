﻿#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"



static bool bbcu_HostOnly = false;


BBCU_DLL_EXPORT int bbcu_GetDeviceCount(void)
{
    int dev_count = 0;
    auto status = hipGetDeviceCount(&dev_count);
    if (status != hipSuccess) {
        dev_count = 0;
    }
    return dev_count;
}

BBCU_DLL_EXPORT int bbcu_GetDevice(void)
{
    int device;
    BB_CUDA_SAFE_CALL(hipGetDevice(&device)); 
    return device;
}

BBCU_DLL_EXPORT void bbcu_SetDevice(int device)
{
    BB_CUDA_SAFE_CALL(hipSetDevice(device)); 
}




BBCU_DLL_EXPORT void bbcu_SetHostOnly(bool hostOnly)
{
    bbcu_HostOnly = hostOnly;
}


BBCU_DLL_EXPORT bool bbcu_IsHostOnly(void)
{
    return bbcu_HostOnly;
}


BBCU_DLL_EXPORT bool bbcu_IsDeviceAvailable(void)
{
    return !bbcu_HostOnly;
}


// end of file
