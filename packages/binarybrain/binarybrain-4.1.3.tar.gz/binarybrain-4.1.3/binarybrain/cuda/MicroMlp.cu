#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"
#include "Common.cuh"


// -------------------------------------------------
//  Forward
// -------------------------------------------------


template <int N=6, int M=16, int MAX_NODE_UNIT>
__global__ void kernal_fp32_MicroMlp_Forward
        (
            float const *x_buf,
            float       *y_buf,
            int   const *input_index,
            float const *hidden_W,
            float const *hidden_b,
            float const *output_W,
            float const *output_b,
            int         node_size,
            int         frame_size,
            int         frame_stride
        )
{
    int const node_id = threadIdx.y;
    int const node    = blockIdx.y * blockDim.y + threadIdx.y;
    int const id      = threadIdx.x;
    int const id_step = blockDim.x;

    // 係数読み込み
    __shared__ float        W0[M][N][MAX_NODE_UNIT];
    __shared__ float        b0[M][MAX_NODE_UNIT];
    __shared__ float        W1[M][MAX_NODE_UNIT];
    __shared__ float        b1[MAX_NODE_UNIT];
    __shared__ float const  *x_ptr[N][MAX_NODE_UNIT];
               float        *y_ptr;

    if ( node < node_size ) {
        for ( int i = id; i < M; i += id_step ) {
            for ( int j = 0; j < N; ++j ) {
                W0[i][j][node_id] = hidden_W[(node * M + i) * N + j];
            }

            b0[i][node_id] = hidden_b[node * M + i];
            W1[i][node_id] = output_W[node * M + i];
        }
        if ( id == 0 ) {
            b1[node_id] = output_b[node];
        }

        // 読み込みアドレス
        for ( int i = 0; i < N; ++i ) {
            int in_idx = input_index[node*N + i];
            x_ptr[i][node_id] = &x_buf[frame_stride * in_idx];
        }

        // 書き込みアドレス
        y_ptr = &y_buf[frame_stride * node];
    }
    
    __syncthreads();
    
    // 1つのSMで1nodeを全フレーム処理
    for ( int frame = id; frame < frame_size; frame += id_step ) {
        if ( node < node_size ) {
            // 入力データ読み込み
            float   x[N];
            for ( int i = 0; i < N; ++i ) {
                x[i] = x_ptr[i][node_id][frame];
            }

            // 計算
            float sig1 = b1[node_id];
            for ( int i = 0; i < M; ++i ) {
                float sig0 = b0[i][node_id];
                for ( int j = 0; j < N; ++j ) {
                    sig0 += x[j] * W0[i][j][node_id];
                }
        
                sig0 = fmaxf(sig0, 0);  // ReLU
        
                sig1 += sig0 * W1[i][node_id];
            }

            // 出力
            y_ptr[frame] = sig1;
        }
        __syncthreads();
    }
}

template <int N=6, int M=16>
int bbcu_fp32_MicroMlp_Forward
        (
            float const     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_hidden_W,
            float const     *dev_hidden_b,
            float const     *dev_output_W,
            float const     *dev_output_b,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            hipStream_t    streamId = 0
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 256;
    unsigned int const MAX_NODE_UNIT  = 16;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

    kernal_fp32_MicroMlp_Forward<N, M, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_hidden_W,
            dev_hidden_b,
            dev_output_W,
            dev_output_b,
            output_node_size,
            frame_size,
            frame_stride
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}

int bbcu_fp32_MicroMlp6x16_Forward
        (
            float const     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_hidden_W,
            float const     *dev_hidden_b,
            float const     *dev_output_W,
            float const     *dev_output_b,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            hipStream_t    streamId
        )
{
    return bbcu_fp32_MicroMlp_Forward<6, 16>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_hidden_W,
            dev_hidden_b,
            dev_output_W,
            dev_output_b,
            input_node_size,
            output_node_size,
            frame_size,
            frame_stride,
            streamId
        );
}



/////////////////////


// bit入力版
template <int N=6, int M=16, int MAX_NODE_UNIT=16>
__global__ void kernal_bit_fp32_MicroMlp_Forward(
            int const       *x_buf,
            float           *y_buf,
            int   const     *input_index,
            float const     *hidden_W,
            float const     *hidden_b,
            float const     *output_W,
            float const     *output_b,
            int             node_size,
            int             frame_size,
            int             input_frame_stride,
            int             output_frame_stride
        )
{
    int const node_id = threadIdx.y;
    int const node    = blockIdx.y * blockDim.y + threadIdx.y;
    int const id      = threadIdx.x;
    int const id_step = blockDim.x;

    // 係数読み込み
    __shared__ float        W0[M][N][MAX_NODE_UNIT];
    __shared__ float        b0[M][MAX_NODE_UNIT];
    __shared__ float        W1[M][MAX_NODE_UNIT];
    __shared__ float        b1[MAX_NODE_UNIT];
    __shared__ int const    *x_ptr[N][MAX_NODE_UNIT];

    if ( node < node_size) {
        for ( int i = id; i < M; i += id_step ) {
            for ( int j = 0; j < N; ++j ) {
                W0[i][j][node_id] = hidden_W[(node * M + i) * N + j];
            }

            b0[i][node_id] = hidden_b[node * M + i];
            W1[i][node_id] = output_W[node * M + i];
        }
        if ( id == 0 ) {
            b1[node_id] = output_b[node];
        }

        // 読み込みアドレス
        for ( int i = 0; i < N; ++i ) {
            int input_node = input_index[node*N + i];
            x_ptr[i][node_id] = &x_buf[input_frame_stride * input_node];
        }
    }
    
    __syncthreads();

    if ( node < node_size) {
        // 書き込みアドレス
        float *y_ptr = &y_buf[output_frame_stride * node];
    
        // 1つのSMで1nodeを全フレーム処理
        for ( int frame = id; frame < frame_size; frame += id_step ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);

            // 入力データ読み込み
            int   x[N];
            for ( int i = 0; i < N; ++i ) {
                x[i] = x_ptr[i][node_id][unit];
            }
        
            // 計算
            float sig1 = b1[node_id];
            for ( int i = 0; i < M; ++i ) {
                float sig0 = b0[i][node_id];
                for ( int j = 0; j < N; ++j ) {
                    if ( x[j] & bit ) {
                        sig0 += W0[i][j][node_id];
                    }
                }
            
                sig0 = fmaxf(sig0, 0);  // ReLU
            
                sig1 += sig0 * W1[i][node_id];
            }

            // 出力
            y_ptr[frame] = sig1;
        }
    }
}


template <int N=6, int M=16>
int bbcu_bit_fp32_MicroMlp_Forward
        (
            int   const     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_hidden_W,
            float const     *dev_hidden_b,
            float const     *dev_output_W,
            float const     *dev_output_b,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             input_frame_stride,
            int             output_frame_stride,
            hipStream_t    streamId = 0
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 512;
    unsigned int const MAX_FRAME_UNIT = 256;
    unsigned int const MAX_NODE_UNIT  = 16;

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

    kernal_bit_fp32_MicroMlp_Forward<N, M, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>
        (
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_hidden_W,
            dev_hidden_b,
            dev_output_W,
            dev_output_b,
            output_node_size,
            frame_size,
            input_frame_stride,
            output_frame_stride
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}


int bbcu_bit_fp32_MicroMlp6x16_Forward
        (
            int   const     *dev_x_buf,
            float           *dev_y_buf,
            int   const     *dev_input_index,
            float const     *dev_hidden_W,
            float const     *dev_hidden_b,
            float const     *dev_output_W,
            float const     *dev_output_b,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             input_frame_stride,
            int             output_frame_stride,
            hipStream_t    streamId
        )
{
    return bbcu_bit_fp32_MicroMlp_Forward<6, 16>
        (
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_hidden_W,
            dev_hidden_b,
            dev_output_W,
            dev_output_b,
            input_node_size,
            output_node_size,
            frame_size,
            input_frame_stride,
            output_frame_stride,
            streamId
        );
}




// -------------------------------------------------
//  Backward
// -------------------------------------------------

#if 0
__device__ __forceinline__ float device_fp32_LocalSum(float v, float *buf)
{
    buf[threadIdx.x] = v;
    __syncthreads();

    // スレッド間集計
    int comb = 1;
    while (comb < blockDim.x) {
        int next = comb * 2;
        int mask = next - 1;
        if ((threadIdx.x & mask) == 0) {
            buf[threadIdx.x] += buf[threadIdx.x + comb];
        }
        comb = next;
        __syncthreads();
    }

    float sum = buf[0];
    __syncthreads();
    
    return sum;
}
#endif


// kernel
template <int N=6, int M=16, int MAX_FRAME_UNIT=32, int MAX_NODE_UNIT=8>
__global__ void kernal_fp32_MicroMlp_Backward
        (
            float const     *x_buf,
            float const     *dy_buf,
            float           *dx_buf,
            int   const     *input_index,
            float const     *hidden_W,
            float const     *hidden_b,
            float           *hidden_dW,
            float           *hidden_db,
            float const     *output_W,
            float const     *output_b,
            float           *output_dW,
            float           *output_db,
            int             node_size,
            int             frame_size,
            int             frame_stride
        )
{
    int const node_id = threadIdx.y;
    int const node    = blockIdx.y * blockDim.y + threadIdx.y;
    int const id      = threadIdx.x;
    int const id_step = blockDim.x;

    __shared__  float       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  float       W0[M][N][MAX_NODE_UNIT];
    __shared__  float       b0[M][MAX_NODE_UNIT];
    __shared__  float       W1[M][MAX_NODE_UNIT];

    __shared__  float       dW0_prev[M][N][MAX_NODE_UNIT];
    __shared__  float       db0_prev[M][MAX_NODE_UNIT];
    __shared__  float       dW1_prev[M][MAX_NODE_UNIT];
    __shared__  float       db1_prev[MAX_NODE_UNIT];

    __shared__  float const *x_ptr[N][MAX_NODE_UNIT];

                float const *dy_ptr;

    if ( node < node_size ) {
        // 係数読み込み
        for ( int i = id; i < M; i += id_step ) {
            for ( int j = 0; j < N; ++j ) {
                W0[i][j][node_id] = hidden_W[(node * M + i) * N + j];
            }

            b0[i][node_id] = hidden_b[node * M + i];
            W1[i][node_id] = output_W[node * M + i];
        }

        // 直前の係数読み込み
        for ( int i = id; i < M; i += id_step ) {
            for ( int j = 0; j < N; ++j ) {
                dW0_prev[i][j][node_id] = hidden_dW[(node * M + i) * N + j];
            }

            db0_prev[i][node_id] = hidden_db[node * M + i];
            dW1_prev[i][node_id] = output_dW[node * M + i];
        }
        if ( id == 0 ) {
            db1_prev[node_id] = output_db[node];
        }

        // ポインタ読み込み
        for ( int i = 0; i < N; ++i ) {
            int input_node = input_index[node*N + i];
            x_ptr[i][node_id] = &x_buf[frame_stride * input_node];
        }

        dy_ptr = &dy_buf[frame_stride * node];
    }

    __syncthreads();
    
    // 勾配初期化
    float dW0[M][N];
    float db0[M];
    float dW1[M];
    float db1;
    for ( int i = 0; i < M; ++ i ) {
        for ( int j = 0; j < N; ++j ) {
            dW0[i][j] = 0;
        }
    }
    for ( int i = 0; i < M; ++i ) {
        db0[i] = 0;
        dW1[i] = 0;
    }
    db1 = 0;
    
    if ( node < node_size ) {
        // 1つのSMで1nodeを全フレーム処理
        for ( int frame = id; frame < frame_size; frame += id_step ) {
            // 入力データ読み込み
            float   x[N];
            for ( int i = 0; i < N; ++i ) {
                x[i] = x_ptr[i][node_id][frame];
            }
        
            // 1段目再計算して2段目逆伝播
            float   grad1 = dy_ptr[frame];
            float   grad0[M];
            db1 += grad1;
            for ( int i = 0; i < M; ++i ) {
                float sig0 = b0[i][node_id];
                for ( int j = 0; j < N; ++j ) {
                    sig0 += x[j] * W0[i][j][node_id];
                }
            
                sig0 = fmaxf(sig0, 0);  // ReLU

                dW1[i] += grad1 * sig0;

                if ( sig0 > 0 ) {       // ReLU
                    grad0[i] = grad1 * W1[i][node_id];
                }
                else {
                    grad0[i] = 0;
                }
            }
        
            // 1段目逆伝播
            float *dx_ptr  = &dx_buf[frame_stride * N * node];
            float   dx[N];
            for ( int i = 0; i < N; ++i ) {
                dx[i] = 0;  // dx_ptr[frame_stride * i + frame];
            }

            for ( int i = 0; i < M; ++i ) {
                db0[i] += grad0[i];
                for ( int j = 0; j < N; ++j ) {
                    dW0[i][j] += grad0[i] * x[j];
                    dx[j]     += grad0[i] * W0[i][j][node_id];
                }
            }
            
            // 誤差書き込み
            for ( int i = 0; i < N; ++i ) {
                dx_ptr[frame_stride * i + frame] = dx[i];
            }
        }
    }
    
    __syncthreads();

    // 係数統合
    for ( int i = 0; i < M; ++i ) {
        for ( int j = 0; j < N; ++j ) {
            dW0[i][j] = device_fp32_LocalSum(dW0[i][j], sbuf[node_id]);
        }
        db0[i] = device_fp32_LocalSum(db0[i], sbuf[node_id]);
        dW1[i] = device_fp32_LocalSum(dW1[i], sbuf[node_id]);
    }
    db1 = device_fp32_LocalSum(db1, sbuf[node_id]);

    // 勾配出力
    if ( node < node_size ) {
        for ( int i = id; i < M; i += id_step ) {
            for ( int j = 0; j < N; ++j ) {
                hidden_dW[(node * M + i) * N + j] = dW0[i][j] + dW0_prev[i][j][node_id];
            }
             hidden_db[node * M + i] = db0[i] + db0_prev[i][node_id];
             output_dW[node * M + i] = dW1[i] + dW1_prev[i][node_id];
        }
        if (id == 0) {
             output_db[node] = db1 + db1_prev[node_id];
        }
    }

    __syncthreads();
}


template <int N=6>
__global__ void kernal_fp32_MicroMlp_BackwardMarge
        (
            float const *src_buf,
            float       *dst_buf,
            int   const *input_index,
            int         node_size,
            int         frame_size,
            int         frame_stride
        )
{
    int frame = blockDim.x * blockIdx.x + threadIdx.x;
    
    for ( int node = 0; node < node_size; ++node ) {
        if ( frame < frame_size ) {
            for ( int n = 0; n < N; ++n ) {
                int in_idx = input_index[node*N + n];
                float*       dst_buf_ptr = &dst_buf[frame_stride * in_idx];
                float        prev_data   = dst_buf_ptr[frame];
                const float* src_buf_ptr = &src_buf[(N * node + n) * frame_stride];
                
                dst_buf_ptr[frame] = prev_data + src_buf_ptr[frame];
            }
        }
        __syncthreads();
    }
}



template <int N=6, int M=16>
int bbcu_fp32_MicroMlp_Backward
        (
            float const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_hidden_W,
            float const     *dev_hidden_b,
            float           *dev_hidden_dW,
            float           *dev_hidden_db,
            float const     *dev_output_W,
            float const     *dev_output_b,
            float           *dev_output_dW,
            float           *dev_output_db,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            hipStream_t    streamId = 0
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        unsigned int const THREAD_SIZE    = 256;
        unsigned int const MAX_FRAME_UNIT = 256;
        unsigned int const MAX_NODE_UNIT  = 16;

#if 0
        dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
        while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
        while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
#else
        dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
        while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
        while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
#endif

        block.x = std::min(block.x, MAX_FRAME_UNIT);
        block.y = std::min(block.y, MAX_NODE_UNIT);
        dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

        kernal_fp32_MicroMlp_Backward<N, M, MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>
            (
                dev_x_buf,
                dev_dy_buf,
                dev_dx_tmp,
                dev_input_index,
                dev_hidden_W,
                dev_hidden_b,
                dev_hidden_dW,
                dev_hidden_db,
                dev_output_W,
                dev_output_b,
                dev_output_dW,
                dev_output_db,
                output_node_size,
                frame_size,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }
    

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

        int block_x = 1024;
        while ( block_x / 2 >= frame_size ) { block_x /= 2; }
        dim3    grid((frame_size + (block_x - 1)) / block_x);
        dim3    block(block_x);
        kernal_fp32_MicroMlp_BackwardMarge<N><<<grid, block>>>
            (
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    return 0;
}


BBCU_DLL_EXPORT int bbcu_fp32_MicroMlp6x16_Backward(
            float const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_hidden_W,
            float const     *dev_hidden_b,
            float           *dev_hidden_dW,
            float           *dev_hidden_db,
            float const     *dev_output_W,
            float const     *dev_output_b,
            float           *dev_output_dW,
            float           *dev_output_db,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             frame_stride,
            hipStream_t    streamId
        )
{
    return bbcu_fp32_MicroMlp_Backward<6, 16>(
            dev_x_buf,
            dev_dy_buf,
            dev_dx_buf,
            dev_dx_tmp,
            dev_input_index,
            dev_hidden_W,
            dev_hidden_b,
            dev_hidden_dW,
            dev_hidden_db,
            dev_output_W,
            dev_output_b,
            dev_output_dW,
            dev_output_db,
            input_node_size,
            output_node_size,
            frame_size,
            frame_stride,
            streamId
        );
}


///////////////////////////////


// kernel
template <int N=6, int M=16, int MAX_FRAME_UNIT=32, int MAX_NODE_UNIT=8>
__global__ void kernal_bit_fp32_MicroMlp_Backward
        (
            int   const     *x_buf,
            float const     *dy_buf,
            float           *dx_buf,
            int   const     *input_index,
            float const     *hidden_W,
            float const     *hidden_b,
            float           *hidden_dW,
            float           *hidden_db,
            float const     *output_W,
            float const     *output_b,
            float           *output_dW,
            float           *output_db,
            int             node_size,
            int             frame_size,
            int             x_frame_stride,
            int             frame_stride
        )
{
    int const node_id = threadIdx.y;
    int const node    = blockIdx.y * blockDim.y + threadIdx.y;
    int const id      = threadIdx.x;
    int const id_step = blockDim.x;

    __shared__  float       sbuf[MAX_NODE_UNIT][MAX_FRAME_UNIT];
    __shared__  float       W0[M][N][MAX_NODE_UNIT];
    __shared__  float       b0[M][MAX_NODE_UNIT];
    __shared__  float       W1[M][MAX_NODE_UNIT];

    __shared__  float       dW0_prev[M][N][MAX_NODE_UNIT];
    __shared__  float       db0_prev[M][MAX_NODE_UNIT];
    __shared__  float       dW1_prev[M][MAX_NODE_UNIT];
    __shared__  float       db1_prev[MAX_NODE_UNIT];

    __shared__  int   const *x_ptr[N][MAX_NODE_UNIT];

                float const *dy_ptr;

    if ( node < node_size ) {
        // 係数読み込み
        for ( int i = id; i < M; i += id_step ) {
            for ( int j = 0; j < N; ++j ) {
                W0[i][j][node_id] = hidden_W[(node * M + i) * N + j];
            }

            b0[i][node_id] = hidden_b[node * M + i];
            W1[i][node_id] = output_W[node * M + i];
        }

        // 直前の係数読み込み
        for ( int i = id; i < M; i += id_step ) {
            for ( int j = 0; j < N; ++j ) {
                dW0_prev[i][j][node_id] = hidden_dW[(node * M + i) * N + j];
            }

            db0_prev[i][node_id] = hidden_db[node * M + i];
            dW1_prev[i][node_id] = output_dW[node * M + i];
        }
        if ( id == 0 ) {
            db1_prev[node_id] = output_db[node];
        }

        // ポインタ読み込み
        for ( int i = 0; i < N; ++i ) {
            int input_node = input_index[node*N + i];
            x_ptr[i][node_id] = &x_buf[x_frame_stride * input_node];
        }

        dy_ptr = &dy_buf[frame_stride * node];
    }

    __syncthreads();
    
    // 勾配初期化
    float dW0[M][N];
    float db0[M];
    float dW1[M];
    float db1;
    for ( int i = 0; i < M; ++ i ) {
        for ( int j = 0; j < N; ++j ) {
            dW0[i][j] = 0;
        }
    }
    for ( int i = 0; i < M; ++i ) {
        db0[i] = 0;
        dW1[i] = 0;
    }
    db1 = 0;
    
    if ( node < node_size ) {
        // 1つのSMで1nodeを全フレーム処理
        for ( int frame = id; frame < frame_size; frame += id_step ) {
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);

            // 入力データ読み込み
            int   x[N];
            for ( int i = 0; i < N; ++i ) {
                x[i] = x_ptr[i][node_id][unit];
            }
            
            // 1段目再計算して2段目逆伝播
            float   grad1 = dy_ptr[frame];
            float   grad0[M];
            db1 += grad1;
            for ( int i = 0; i < M; ++i ) {
                float sig0 = b0[i][node_id];
                for ( int j = 0; j < N; ++j ) {
                    if ( x[j] & bit ) {
                        sig0 += W0[i][j][node_id];
                    }
                }
            
                sig0 = fmaxf(sig0, 0);  // ReLU

                dW1[i] += grad1 * sig0;

                if ( sig0 > 0 ) {       // ReLU
                    grad0[i] = grad1 * W1[i][node_id];
                }
                else {
                    grad0[i] = 0;
                }
            }
        
            // 1段目逆伝播
            float *dx_ptr  = &dx_buf[frame_stride * N * node];
            float   dx[N];
            for ( int i = 0; i < N; ++i ) {
                dx[i] = 0;  // dx_ptr[frame_stride * i + frame];
            }

            for ( int i = 0; i < M; ++i ) {
                db0[i] += grad0[i];
                for ( int j = 0; j < N; ++j ) {
                    if ( x[j] & bit ) { dW0[i][j] += grad0[i]; }
                    dx[j]     += grad0[i] * W0[i][j][node_id];
                }
            }
            
            // 誤差書き込み
            for ( int i = 0; i < N; ++i ) {
                dx_ptr[frame_stride * i + frame] = dx[i];
            }
        }
    }
    
    __syncthreads();

    // 係数統合
    for ( int i = 0; i < M; ++i ) {
        for ( int j = 0; j < N; ++j ) {
            dW0[i][j] = device_fp32_LocalSum(dW0[i][j], sbuf[node_id]);
        }
        db0[i] = device_fp32_LocalSum(db0[i], sbuf[node_id]);
        dW1[i] = device_fp32_LocalSum(dW1[i], sbuf[node_id]);
    }
    db1 = device_fp32_LocalSum(db1, sbuf[node_id]);

    // 勾配出力
    if ( node < node_size ) {
        for ( int i = id; i < M; i += id_step ) {
            for ( int j = 0; j < N; ++j ) {
                hidden_dW[(node * M + i) * N + j] = dW0[i][j] + dW0_prev[i][j][node_id];
            }
             hidden_db[node * M + i] = db0[i] + db0_prev[i][node_id];
             output_dW[node * M + i] = dW1[i] + dW1_prev[i][node_id];
        }
        if (id == 0) {
             output_db[node] = db1 + db1_prev[node_id];
        }
    }

    __syncthreads();
}


template <int N=6, int M=16>
int bbcu_bit_fp32_MicroMlp_Backward
        (
            int   const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_hidden_W,
            float const     *dev_hidden_b,
            float           *dev_hidden_dW,
            float           *dev_hidden_db,
            float const     *dev_output_W,
            float const     *dev_output_b,
            float           *dev_output_dW,
            float           *dev_output_db,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             x_frame_stride,
            int             frame_stride,
            hipStream_t    streamId = 0
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    {
        unsigned int const THREAD_SIZE    = 256;
        unsigned int const MAX_FRAME_UNIT = 256;
        unsigned int const MAX_NODE_UNIT  = 16;

#if 0
        dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
        while ( (int)block.x / 2 >= frame_size )       { block.x /= 2; block.y *= 2; }
        while ( (int)block.y / 2 >= output_node_size ) { block.y /= 2; }
#else
        dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
        while ( (int)block.y / 2 >= output_node_size) { block.y /= 2; block.x *= 2;}
        while ( (int)block.x / 2 >= frame_size      ) { block.x /= 2; }
#endif

        block.x = std::min(block.x, MAX_FRAME_UNIT);
        block.y = std::min(block.y, MAX_NODE_UNIT);
        dim3    grid(1, (output_node_size + (block.y - 1)) / block.y);

        kernal_bit_fp32_MicroMlp_Backward<N, M, MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>
            (
                dev_x_buf,
                dev_dy_buf,
                dev_dx_tmp,
                dev_input_index,
                dev_hidden_W,
                dev_hidden_b,
                dev_hidden_dW,
                dev_hidden_db,
                dev_output_W,
                dev_output_b,
                dev_output_dW,
                dev_output_db,
                output_node_size,
                frame_size,
                x_frame_stride,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }
    

    {
        BB_CUDA_SAFE_CALL(hipMemset(dev_dx_buf, 0, input_node_size * frame_stride * sizeof(float)));

        int block_x = 1024;
        while ( block_x / 2 >= frame_size ) { block_x /= 2; }
        dim3    grid((frame_size + (block_x - 1)) / block_x);
        dim3    block(block_x);
        kernal_fp32_MicroMlp_BackwardMarge<N><<<grid, block>>>
            (
                dev_dx_tmp,
                dev_dx_buf,
                dev_input_index,
                output_node_size,
                frame_size,
                frame_stride
            );
        BB_CUDA_CHECK_LAST_ERROR();
    }

    return 0;
}


BBCU_DLL_EXPORT int bbcu_bit_fp32_MicroMlp6x16_Backward
        (
            int   const     *dev_x_buf,
            float const     *dev_dy_buf,
            float           *dev_dx_buf,
            float           *dev_dx_tmp,
            int   const     *dev_input_index,
            float const     *dev_hidden_W,
            float const     *dev_hidden_b,
            float           *dev_hidden_dW,
            float           *dev_hidden_db,
            float const     *dev_output_W,
            float const     *dev_output_b,
            float           *dev_output_dW,
            float           *dev_output_db,
            int             input_node_size,
            int             output_node_size,
            int             frame_size,
            int             x_frame_stride,
            int             frame_stride,
            hipStream_t    streamId
        )
{
    return bbcu_bit_fp32_MicroMlp_Backward<6, 16>
        (
            dev_x_buf,
            dev_dy_buf,
            dev_dx_buf,
            dev_dx_tmp,
            dev_input_index,
            dev_hidden_W,
            dev_hidden_b,
            dev_hidden_dW,
            dev_hidden_db,
            dev_output_W,
            dev_output_b,
            dev_output_dW,
            dev_output_db,
            input_node_size,
            output_node_size,
            frame_size,
            x_frame_stride,
            frame_stride,
            streamId
        );
}



// end of file
