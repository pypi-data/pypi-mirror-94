#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 NVIDIA Corporation.
 * Copyright (c) 2018-2020 Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "gpu.cuh"
#include "math_functions.cuh"

#include <hipsparse.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAUtils.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <torch/extension.h>
#include <torch/script.h>

namespace minkowski {

template <typename Itype, typename Dtype>
__global__ void
unique_row2num_nonzero(const int n, Dtype *__restrict__ d_num_nonzero,
                       const Itype *__restrict__ unique_row_ptr,
                       const Dtype *__restrict__ reduced_val_ptr) {
  CUDA_KERNEL_LOOP(index, n) {
    d_num_nonzero[unique_row_ptr[index]] = reduced_val_ptr[index];
  }
}

hipDataType getTensorCudaDataType(torch::Tensor const &self) {
  hipDataType cuda_data_type;
  switch (self.scalar_type()) {
  case torch::ScalarType::Float:
    cuda_data_type = HIP_R_32F;
    break;
  case torch::ScalarType::Double:
    cuda_data_type = HIP_R_64F;
    break;
  default:
    TORCH_CHECK(false, "Tensor types must be either float32 or float64");
    break;
  }
  return cuda_data_type;
}

template <typename th_int_type>
std::pair<torch::Tensor, torch::Tensor>
coo_spmm(torch::Tensor const &rows, torch::Tensor const &cols,
         torch::Tensor const &vals, int64_t const dim_i, int64_t const dim_j,
         torch::Tensor const &mat2, int64_t const spmm_algorithm_id,
         bool const return_num_nonzero) {
#if defined __HIP_PLATFORM_HCC__
  TORCH_CHECK(false, "spmm sparse-dense is not supported on HIP");
#elif defined(_WIN32) || defined(_WIN64)
  TORCH_CHECK(false, "spmm sparse-dense CUDA is not supported on Windows");
#elif !defined(CUDART_VERSION)
  TORCH_CHECK(false, "CUDART_VERSION not defined");
#endif

  constexpr bool is_int32 = std::is_same<th_int_type, int32_t>::value;
  constexpr bool is_int64 = std::is_same<th_int_type, int64_t>::value;

  hipsparseSpMMAlg_t mm_alg;
#if defined(CUDART_VERSION) && (CUDART_VERSION < 10010)
  TORCH_CHECK(false, "spmm sparse-dense requires CUDA 10.1 or greater");
#elif defined(CUDART_VERSION) && (CUDART_VERSION >= 10010) &&                  \
    (CUDART_VERSION < 11000)
  switch (spmm_algorithm_id) {
  case 1:
    mm_alg = HIPSPARSE_COOMM_ALG1;
    break;
  case 2:
    mm_alg = HIPSPARSE_COOMM_ALG2;
    break;
  case 3:
    mm_alg = HIPSPARSE_COOMM_ALG3;
    break;
  default:
    TORCH_CHECK(false, "Invalid algorithm id.", spmm_algorithm_id);
    mm_alg = HIPSPARSE_MM_ALG_DEFAULT;
  }
  TORCH_CHECK(is_int32, "int64 hipsparseSpMM requires CUDA 11.1 or greater");
#elif defined(CUDART_VERSION) && (CUDART_VERSION >= 11000)
  switch (spmm_algorithm_id) {
  case 1:
    mm_alg = HIPSPARSE_SPMM_COO_ALG1;
    break;
  case 2:
    mm_alg = HIPSPARSE_SPMM_COO_ALG2;
    break;
  case 3:
    mm_alg = HIPSPARSE_SPMM_COO_ALG3;
    break;
  case 4:
    mm_alg = HIPSPARSE_SPMM_COO_ALG4;
    break;
  default:
    TORCH_CHECK(false, "Invalid algorithm id.", spmm_algorithm_id);
    mm_alg = HIPSPARSE_SPMM_ALG_DEFAULT;
  }
  TORCH_CHECK(is_int32, "int64 coosort not implemented");
  // coosort not supported with int64 || (is_int64 && (mm_alg ==
  // HIPSPARSE_SPMM_COO_ALG4)));
#endif

  at::ScalarType int_scalar_type = std::is_same<th_int_type, int32_t>::value
                                       ? at::ScalarType::Int
                                       : at::ScalarType::Long;

  ASSERT(rows.scalar_type() == int_scalar_type, "int type mismatch.");

  ASSERT(rows.scalar_type() == cols.scalar_type(),
         "rows and cols must have the same scalar type.");
  ASSERT(rows.scalar_type() == cols.scalar_type(),
         "rows and cols must have the same scalar type.");
  ASSERT(vals.scalar_type() == mat2.scalar_type(),
         "vals and mat2 must have the same scalar type.");

  ASSERT(rows.is_contiguous(), "rows must be contiguous");
  ASSERT(cols.is_contiguous(), "cols must be contiguous");
  ASSERT(vals.is_contiguous(), "vals must be contiguous");
  ASSERT(mat2.is_contiguous(), "mat2 must be contiguous");

  ASSERT(rows.is_cuda(), "rows must be CUDA, but got CPU");
  ASSERT(cols.is_cuda(), "cols must be CUDA, but got CPU");
  ASSERT(vals.is_cuda(), "vals must be CUDA, but got CPU");
  ASSERT(mat2.is_cuda(), "mat2 must be CUDA, but got CPU");
  ASSERT(at::cuda::check_device({rows, cols, vals, mat2}),
         "All inputs must be on the same device.");

  ASSERT(mat2.dim() == 2, "Tensor 'mat2' must have 2 dims, but has ",
         mat2.dim());

  // int64_t dim_i = self.size(0);
  // int64_t dim_j = self.size(1);
  int64_t dim_k = mat2.size(1);

  torch::Tensor result = at::zeros({dim_k, dim_i}, mat2.options());
  torch::Tensor num_nonzero = at::zeros({0}, mat2.options());

  // Create tensors to view just the current set of matrices
  int64_t const nnz = rows.numel();

  if ((dim_j == 0) || (dim_k == 0) || (nnz == 0)) {
    return std::make_pair(result, num_nonzero);
  }

  // Dense matrices have to be contiguous for hipsparseSpMM to work
  torch::Tensor const mat2_contig = mat2.contiguous();
  auto cusparse_handle = at::cuda::getCurrentCUDASparseHandle();

  torch::Scalar beta = 0;
  torch::Scalar alpha = 1;

  hipDataType cuda_data_type = getTensorCudaDataType(mat2_contig);
  th_int_type *row_indices_ptr =
      reinterpret_cast<th_int_type *>(rows.data_ptr());
  th_int_type *col_indices_ptr =
      reinterpret_cast<th_int_type *>(cols.data_ptr());

  // Iterate through each set of 2D matrices within the 3D
  // tensor inputs, performing a matrix multiply with each
  AT_DISPATCH_FLOATING_TYPES(vals.scalar_type(), "coo_spmm", [&] {
    scalar_t alpha_val = alpha.to<scalar_t>();
    scalar_t beta_val = beta.to<scalar_t>();

    scalar_t *values_ptr = reinterpret_cast<scalar_t *>(vals.data_ptr());
    scalar_t *mat2_ptr = reinterpret_cast<scalar_t *>(mat2_contig.data_ptr());
    scalar_t *result_ptr = reinterpret_cast<scalar_t *>(result.data_ptr());

    //////////////////////////////////////
    // Sort the sparse matrix COO
    th_int_type *sorted_row_ptr =
        (th_int_type *)c10::cuda::CUDACachingAllocator::raw_alloc(
            2 * (nnz + 1) * sizeof(th_int_type));
    th_int_type *sorted_col_ptr = sorted_row_ptr + nnz + 1;
    scalar_t *sorted_val_ptr =
        (scalar_t *)c10::cuda::CUDACachingAllocator::raw_alloc(
            nnz * sizeof(scalar_t));
    LOG_DEBUG("Allocated sorted row col val", nnz);

    // Copy the indices
    CUDA_CHECK(hipMemcpy(sorted_row_ptr, row_indices_ptr,
                          nnz * sizeof(th_int_type), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(sorted_col_ptr, col_indices_ptr,
                          nnz * sizeof(th_int_type), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(sorted_val_ptr, values_ptr, nnz * sizeof(scalar_t),
                          hipMemcpyDeviceToDevice));

    thrust::sort_by_key(thrust::device,            //
                        sorted_row_ptr,            // key begin
                        sorted_row_ptr + nnz,      // key end
                        thrust::make_zip_iterator( // value begin
                            thrust::make_tuple(    //
                                sorted_col_ptr,    //
                                sorted_val_ptr     //
                                )                  //
                            ));
    LOG_DEBUG("Sorted row");
    //////////////////////////////////////

    size_t workspace_buffer_size = 0;
    void *workspace_buffer = nullptr;

    hipsparseSpMatDescr_t sparse_descr;
    CUSPARSE_CHECK(hipsparseCreateCoo(
        &sparse_descr,     //
        dim_i, dim_j, nnz, //
        reinterpret_cast<void *>(sorted_row_ptr),
        reinterpret_cast<void *>(sorted_col_ptr),
        reinterpret_cast<void *>(sorted_val_ptr), //
        std::is_same<th_int_type, int32_t>::value ? HIPSPARSE_INDEX_32I
                                                  : HIPSPARSE_INDEX_64I,
        HIPSPARSE_INDEX_BASE_ZERO, cuda_data_type));

    hipsparseDnMatDescr_t dense_descr;
    CUSPARSE_CHECK(hipsparseCreateDnMat(&dense_descr,                       //
                                       dim_k, dim_j, dim_k,                //
                                       reinterpret_cast<void *>(mat2_ptr), //
                                       cuda_data_type, HIPSPARSE_ORDER_COL));

    hipsparseDnMatDescr_t result_descr;
    CUSPARSE_CHECK(hipsparseCreateDnMat(&result_descr,                        //
                                       dim_i, dim_k, dim_i,                  //
                                       reinterpret_cast<void *>(result_ptr), //
                                       cuda_data_type, HIPSPARSE_ORDER_COL));

    size_t required_workspace_buffer_size = 0;
    CUSPARSE_CHECK(hipsparseSpMM_bufferSize(
        cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_TRANSPOSE, (void *)&alpha_val, sparse_descr,
        dense_descr, (void *)&beta_val, result_descr, cuda_data_type, mm_alg,
        &required_workspace_buffer_size));
    LOG_DEBUG("Buffer size:", required_workspace_buffer_size);

    if (required_workspace_buffer_size > workspace_buffer_size) {
      if (workspace_buffer != nullptr) {
        hipFree(workspace_buffer);
      }
      workspace_buffer_size = required_workspace_buffer_size;
      LOG_DEBUG("hipMallocManaged");
      hipMallocManaged(&workspace_buffer, workspace_buffer_size);
    }
    LOG_DEBUG("SPMM");
    CUSPARSE_CHECK(hipsparseSpMM(cusparse_handle,                  //
                                HIPSPARSE_OPERATION_NON_TRANSPOSE, //
                                HIPSPARSE_OPERATION_TRANSPOSE,     //
                                (void *)&alpha_val,               //
                                sparse_descr, dense_descr,        //
                                (void *)&beta_val, result_descr,  //
                                cuda_data_type, mm_alg, workspace_buffer));
    CUSPARSE_CHECK(hipsparseDestroySpMat(sparse_descr));
    CUSPARSE_CHECK(hipsparseDestroyDnMat(dense_descr));
    CUSPARSE_CHECK(hipsparseDestroyDnMat(result_descr));

    // Num nonzer
    if (return_num_nonzero) {
      th_int_type *unique_row_ptr =
          (th_int_type *)c10::cuda::CUDACachingAllocator::raw_alloc(
              nnz * sizeof(th_int_type));
      scalar_t *reduced_val_ptr =
          (scalar_t *)c10::cuda::CUDACachingAllocator::raw_alloc(
              nnz * sizeof(scalar_t));
      torch::Tensor ones = at::zeros({nnz}, mat2.options());

      num_nonzero.resize_({dim_i, 1});
      num_nonzero.zero_();

      // reduce by key
      auto end = thrust::reduce_by_key(
          thrust::device,                                // policy
          sorted_row_ptr,                                // key begin
          sorted_row_ptr + nnz,                          // key end
          reinterpret_cast<scalar_t *>(ones.data_ptr()), // value begin
          unique_row_ptr,                                // key out begin
          reduced_val_ptr                                // value out begin
      );

      int num_unique_keys = end.first - unique_row_ptr;
      LOG_DEBUG("Num unique keys:", num_unique_keys);

      // Copy the results to the correct output
      unique_row2num_nonzero<th_int_type, scalar_t>
          <<<GET_BLOCKS(num_unique_keys, 128), 128>>>(
              num_unique_keys,
              reinterpret_cast<scalar_t *>(num_nonzero.data_ptr()),
              unique_row_ptr, reduced_val_ptr);

      c10::cuda::CUDACachingAllocator::raw_delete((void *)unique_row_ptr);
      c10::cuda::CUDACachingAllocator::raw_delete((void *)reduced_val_ptr);
    }

    LOG_DEBUG("Dealloc");
    c10::cuda::CUDACachingAllocator::raw_delete((void *)sorted_row_ptr);
    c10::cuda::CUDACachingAllocator::raw_delete((void *)sorted_val_ptr);

    if (workspace_buffer != nullptr) {
      hipFree(workspace_buffer);
    }
  });

  // Need to transpose the result matrices since cusparse stores
  // them in column-major order in memory
  result.transpose_(0, 1);

  CUDA_CHECK(hipGetLastError());

  return std::make_pair(result, num_nonzero);
}

template std::pair<torch::Tensor, torch::Tensor>
coo_spmm<int32_t>(torch::Tensor const &rows, torch::Tensor const &cols,
                  torch::Tensor const &vals, int64_t const dim_i,
                  int64_t const dim_j, torch::Tensor const &mat2,
                  int64_t const spmm_algorithm_id,
                  bool const return_num_nonzero);

// template torch::Tensor
// coo_spmm<int64_t>(torch::Tensor const &rows, torch::Tensor const &cols,
//                   torch::Tensor const &vals, int64_t const dim_i,
//                   int64_t const dim_j, torch::Tensor const &mat2,
//                   int64_t spmm_algorithm_id);

} // namespace minkowski
