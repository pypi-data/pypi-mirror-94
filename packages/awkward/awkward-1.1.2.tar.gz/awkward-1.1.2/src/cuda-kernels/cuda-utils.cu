// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

#define FILENAME(line) FILENAME_FOR_EXCEPTIONS_CUDA("src/cuda-kernels/cuda-utils.cu", line)

#include "awkward/cuda-utils.h"

ERROR awkward_cuda_ptr_device_num(int64_t* num, void* ptr) {
  hipPointerAttribute_t att;
  hipError_t status = hipPointerGetAttributes(&att, ptr);
  if (status != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(status), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  *num = att.device;
  return success();
}

ERROR awkward_cuda_ptr_device_name(char* name, void* ptr) {
  hipPointerAttribute_t att;
  hipError_t status = hipPointerGetAttributes(&att, ptr);
  if (status != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(status), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  hipDeviceProp_t dev_prop;
  status = hipGetDeviceProperties(&dev_prop, att.device);
  if (status != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(status), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  strcpy(name, dev_prop.name);
  return success();
}

ERROR awkward_cuda_host_to_device(
  void* to_ptr,
  void* from_ptr,
  int64_t bytelength) {
  hipError_t memcpy_stat = hipMemcpy(
    to_ptr, from_ptr, bytelength, hipMemcpyHostToDevice);
  if (memcpy_stat != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(memcpy_stat), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  else {
    return success();
  }
}

ERROR awkward_cuda_device_to_host(
  void* to_ptr,
  void* from_ptr,
  int64_t bytelength) {
  hipError_t memcpy_stat = hipMemcpy(to_ptr,
                                       from_ptr,
                                       bytelength,
                                       hipMemcpyDeviceToHost);
  if (memcpy_stat != hipError_t::hipSuccess) {
    return failure_pass_through(hipGetErrorString(memcpy_stat), kSliceNone, kSliceNone, FILENAME(__LINE__));
  }
  else {
    return success();
  }
}
